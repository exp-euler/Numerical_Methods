#include "hip/hip_runtime.h"
#include "Matrix.hpp"
#include "Vector.hpp"
#include "MatrixFunctions.hpp"
#include <ctime>

#ifdef GPU
#include <hip/driver_types.h>
__global__ void MM1 (float *a, float *b, float *c, int n)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    float temp = 0;
    for(int k=0; k<n;k++)
        temp += a[i*n+k] * b[k*n+j];
    c[i*n+j] = temp;
}
#endif

int main(){
    srand(static_cast<unsigned>(time(0)));
    int size = 16*16;
    int rand_range = 2;

    // Objects for the scalar CPU multiplication
    Matrix<float> a(size,size);
    Matrix<float> b(size,size);
    Matrix<float> c(size,size);

    // Objects for the GPU multiplication
    float *A, *B, *C;           // host copies
    float *d_A, *d_B, *d_C;     // device copies
    size_t mem_size = size * size * sizeof(float);

    hipMalloc((void **)&d_A, mem_size);
    hipMalloc((void **)&d_B, mem_size);
    hipMalloc((void **)&d_C, mem_size);

    A = (float *)malloc(mem_size);
    B = (float *)malloc(mem_size);
    C = (float *)malloc(mem_size);

    float numA;
    float numB;
    for(int i=0; i<size; i++) {
        for(int j=0; j<size; j++) {
            numA = 1 + (rand() % rand_range);
            numB = 2 + (rand() % rand_range);
            a(i,j) = numA;
            b(i,j) = numB;
            A[i*size+j] = numA;
            B[i*size+j] = numB;
        }
    }

    // Copy inputs to device
    hipMemcpy(d_A, A, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, mem_size, hipMemcpyHostToDevice);

    // GPU multiplication
    int p=16; int q=16;
    dim3 grid (size/q, size/p);
    dim3 block (q,p);
    MM1 <<<grid, block>>> (d_A, d_B, d_C, size);

    hipMemcpy(C, d_C, mem_size, hipMemcpyDeviceToHost);

    // Serial CPU multiplication
    c = a*b;

    // Compare results
    for(int i=0; i<size; i++) {
        for(int j=0; j<size; j++) {
            if(C[i*size+j] - c(i,j) != 0) {
                std::cout << "Results differ" << std::endl;
                // Cleanup
                free(A); free(B); free(C);
                hipFree(d_A); hipFree(d_B); hipFree(d_C);
            }
            return 0;
        }
    }


    // Cleanup
    free(A); free(B); free(C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
